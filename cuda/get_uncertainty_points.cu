#include "hip/hip_runtime.h"
#include "include/common.h"
#include "iostream"

using Vec2ida = Eigen::Matrix<int, 2, 1, Eigen::DontAlign>;
using Vec6ida = Eigen::Matrix<int, 6, 1, Eigen::DontAlign>;
using Vec3ida = Eigen::Matrix<int, 3, 1, Eigen::DontAlign>;

namespace kinectfusion {
    namespace internal {
        namespace cuda {

            __global__
            void update_uncertainty_kernel(
                                    PtrStepSz<short> uncertainty_volume,
                                    int3 volume_size, 
                                    const Vec3fda camera_pos)
            {
                const int x = blockIdx.x * blockDim.x + threadIdx.x;
                const int y = blockIdx.y * blockDim.y + threadIdx.y;

                if (x >= volume_size.x || y >= volume_size.y)
                    return;

                for (int z = 0; z < volume_size.z; ++z) {
                    if(uncertainty_volume.ptr(z * volume_size.y + y)[x] !=2)
                        continue;

                    //uncertainty_volume.ptr(z * volume_size.y + y)[x]  = -2;
                    //射线求线段穿过的体素块
                   Vec3fda startPt(float(x)+0.5,float(y)+0.5,float(z)+0.5);
                   Vec3fda dir = camera_pos - startPt;

                   //根据方向获得可能相交的三个面
                   int xi, yi ,zi , nx, ny, nz;
                   xi = 0; yi=1; zi=2; nx = ny = nz = -1;

                   if(dir.x()>0)
                   {
                        xi = 3;
                        nx = 1;
                   }

                   if(dir.y()>0)
                   {
                        yi = 4;
                        ny = 1;
                   }

                    if(dir.z()>0)
                   {
                        zi = 5;
                        nz = 1;
                   }
             
                   //三维DDA
                   Vec6ida  currBox;
                   currBox<<x, y, z, x+1, y+1, z+1;
                   Vec3fda  currPt = startPt;

                   float tsum = 0;//用于终止
                   int  boxCount = 0;
                   do{

                        if (boxCount>4)
                            uncertainty_volume.ptr(currBox[2]* volume_size.y  +currBox[1] )[currBox[0]] = 0;
                        boxCount++;

                        float tx = abs((currBox[xi] - currPt.x())/dir.x());
                        float ty = abs((currBox[yi] - currPt.y())/dir.y());
                        float tz = abs((currBox[zi] - currPt.z())/dir.z());

                        int tnx = nx;
                        int tny = ny;
                        int tnz = nz;
                    
                        float t = tx ;
                        if(ty < tx )
                        {
                            tnx = 0;
                            t = ty;
                        }
                        else
                            tny = 0;

                        if(tz < t)
                        {
                            tnx = tny =0;
                            t = tz;
                        }
                        else
                            tnz = 0;
                         

                        currPt = currPt + dir*(t);

                        currBox[0] = currBox[0] + tnx;
                        currBox[1] = currBox[1] + tny;
                        currBox[2] =  currBox[2] + tnz;
                        currBox[3] =  currBox[3] + tnx;
                        currBox[4] =  currBox[4] + tny;
                        currBox[5] =  currBox[5] + tnz;
                        
                        tsum += t;

                        if(tsum>1.0)
                            break;
                        
                   }while(1);
                }
            }

              void uncertainty_map(
                                        VolumeData& volume,
                                        const Eigen::Matrix4f& model_view
            )
            {
                Eigen::Matrix<float, 3, 3, Eigen::DontAlign> rotation =   model_view.block(0, 0, 3, 3);
                Vec3fda translation = model_view.block(0, 3, 3, 1);
                const Vec3fda camera_pos = (rotation * Vec3fda(0,0,0)  + translation)/ volume.voxel_scale;
                const Vec3ida camera_pos_int(int(camera_pos.x()), int(camera_pos.y()), int(camera_pos.z()));

                const dim3 threads(32, 32);
                const dim3 blocks((volume.volume_size.x + threads.x - 1) / threads.x,
                                  (volume.volume_size.y + threads.y - 1) / threads.y);
                                  
                update_uncertainty_kernel<<<blocks, threads>>>(
                    volume.uncertainty_volume,
                    volume.volume_size, 
                    camera_pos
                );
                
                hipDeviceSynchronize();
            }    
            
            __global__
            void update_uncertainty_empty_kernel(
                                    const PtrStepSz<float> depth_image,
                                    const float depth_cutoff_distance,
                                    CameraParameters cam_params,
                                    PtrStepSz<short> uncertainty_volume,
                                    int3 volume_size, 
                                    const Vec3fda camera_pos,
                                    Eigen::Matrix<float, 3, 3, Eigen::DontAlign> rotation, 
                                    Vec3fda translation, const float voxel_scale)
            {
                const int u = blockIdx.x * blockDim.x + threadIdx.x;
                const int v = blockIdx.y * blockDim.y + threadIdx.y;

                if (u >= depth_image.cols || v >= depth_image.rows)
                    return;

                if(depth_image.ptr(v)[u]!=0)
                    return;
                
               Vec3fda xylambda(
                            (u- cam_params.principal_x) / cam_params.focal_x,
                            (v- cam_params.principal_y) / cam_params.focal_y,
                            1.f);
                
                xylambda.normalize();

                xylambda = xylambda * depth_cutoff_distance;
                Vec3fda startPt = (rotation*(xylambda)+translation)/voxel_scale;

                int x = int(startPt.x());
                int y = int(startPt.y());
                int z = int(startPt.z());


                //射线求线段穿过的体素块
               
                Vec3fda dir = camera_pos - startPt;

                //根据方向获得可能相交的三个面
                int xi, yi ,zi , nx, ny, nz;
                xi = 0; yi=1; zi=2; nx = ny = nz = -1;

                if(dir.x()>0)
                {
                        xi = 3;
                        nx = 1;
                 }

                if(dir.y()>0)
                {
                    yi = 4;
                    ny = 1;
                 }

                if(dir.z()>0)
                {
                    zi = 5;
                    nz = 1;
                }
             
                //三维DDA 
                Vec6ida  currBox;
                currBox<<x, y, z, x+1, y+1, z+1;
                Vec3fda  currPt = startPt;

                float tsum = 0;//用于终止
                do{
                    
                    if(currBox[0]>=0&&currBox[1]>=0&&currBox[2]>=0&&
                        currBox[0]<volume_size.x&&currBox[1]<volume_size.y&&currBox[2]<volume_size.z)
                   {
                        if(uncertainty_volume.ptr(currBox[2]* volume_size.y  +currBox[1] )[currBox[0]] == -1 )
                            uncertainty_volume.ptr(currBox[2]* volume_size.y  +currBox[1] )[currBox[0]] = 0;
                    }

                    float tx = abs((currBox[xi] - currPt.x())/dir.x());
                    float ty = abs((currBox[yi] - currPt.y())/dir.y());
                    float tz = abs((currBox[zi] - currPt.z())/dir.z());

                    int tnx = nx;
                    int tny = ny;
                    int tnz = nz;
                    
                    float t = tx ;
                    if(ty < tx )
                    {
                        tnx = 0;
                        t = ty;
                    }
                    else
                        tny = 0;

                    if(tz < t)
                    {
                        tnx = tny =0;
                        t = tz;
                    }
                    else
                        tnz = 0;
                         
                    currPt = currPt + dir*(t);

                    currBox[0] = currBox[0] + tnx;
                    currBox[1] = currBox[1] + tny;
                    currBox[2] =  currBox[2] + tnz;
                    currBox[3] =  currBox[3] + tnx;
                    currBox[4] =  currBox[4] + tny;
                    currBox[5] =  currBox[5] + tnz;
                        
                    tsum += t;

                    if(tsum>1.0)  
                        break;
                        
                }while(1);
            }

            void uncertainty_map_empty (const cv::cuda::GpuMat& depth_image, 
                                                                           const float depth_cutoff_distance,
                                                                           const CameraParameters& camera_params,
                                                                           VolumeData& volume,
                                                                           const Eigen::Matrix4f& model_view)
            {
                Eigen::Matrix<float, 3, 3, Eigen::DontAlign> rotation =   model_view.block(0, 0, 3, 3);
                Vec3fda translation = model_view.block(0, 3, 3, 1);
                const Vec3fda camera_pos = (rotation * Vec3fda(0,0,0)  + translation)/ volume.voxel_scale;

                const dim3 threads(32, 32);
                const dim3 blocks((camera_params.image_width + threads.x - 1) / threads.x,
                                  (camera_params.image_height + threads.y - 1) / threads.y);
                
                update_uncertainty_empty_kernel<<<blocks, threads>>>(
                    depth_image, 
                    depth_cutoff_distance,
                    camera_params,
                    volume.uncertainty_volume,
                    volume.volume_size, 
                    camera_pos,
                    model_view.block(0, 0, 3, 3),
                    model_view.block(0, 3, 3, 1),
                    volume.voxel_scale
                );         
                hipDeviceSynchronize();     
            }  

              __global__
            void  extend_mesh_voxel(
                PtrStepSz<short> uncertainty_volume,
                PtrStepSz<short3> offsetMat,
                int offset_size,
                int3 volume_size
            )
            {
                const int x = blockIdx.x * blockDim.x + threadIdx.x;
                const int y = blockIdx.y * blockDim.y + threadIdx.y;

                if (x >= volume_size.x || y >= volume_size.y)
                    return;

                 for(int z =0; z < volume_size.z; ++z )
                {
                      if(uncertainty_volume.ptr(z * volume_size.y + y)[x]!=1)
                        continue;

                    for(int i=0; i < offset_size; ++i)
                    {
                        int tx = x + offsetMat.ptr(0)[i].x;
                        int ty = y + offsetMat.ptr(0)[i].y;
                        int tz = z + offsetMat.ptr(0)[i].z;

                        if(tx >= volume_size.x || ty >= volume_size.y|| tz >= volume_size.z||
                            tx<0 || ty<0 || tz<0 )
                            continue;

                        if(uncertainty_volume.ptr(tz * volume_size.y + ty)[tx]==0||
                            uncertainty_volume.ptr(tz * volume_size.y + ty)[tx]==-1)
                            uncertainty_volume.ptr(tz * volume_size.y + ty)[tx]=2;
                    }
                }
            }

            __global__
            void  mark_uncertainty_voxels(
                PtrStepSz<short> uncertainty_volume,
                PtrStepSz<short3> offsetMat,
                int offset_size,
                int3 volume_size
            )
            {
                const int x = blockIdx.x * blockDim.x + threadIdx.x;
                const int y = blockIdx.y * blockDim.y + threadIdx.y;

                if (x >= volume_size.x || y >= volume_size.y)
                    return;

                for(int z =0; z < volume_size.z; ++z )
                {
                    if(uncertainty_volume.ptr(z * volume_size.y + y)[x]!=2)
                        continue;
                        
                    //扩散查找
                    int value = 0;
                    bool hasUnKown = false;
                    for(int i=0; i < offset_size; ++i)
                    {
                        int tx = x + offsetMat.ptr(0)[i].x;
                        int ty = y + offsetMat.ptr(0)[i].y;
                        int tz = z + offsetMat.ptr(0)[i].z;

                        if(tx >= volume_size.x || ty >= volume_size.y || tz >= volume_size.z||
                            tx<0 || ty<0 || tz<0 )
                            continue;

                        if(uncertainty_volume.ptr(tz * volume_size.y + ty)[tx] >=0)
                            value |=  (1<<uncertainty_volume.ptr(tz * volume_size.y + ty)[tx]);
                        else
                            hasUnKown = true;
                    }

                    if((value != 5)||(!hasUnKown))
                        continue;
                    uncertainty_volume.ptr(z * volume_size.y + y)[x] = 3;
                }
            }

            std::vector<Eigen::Vector3f>  get_uncertainty_points_cuda( const VolumeData& volume, 
                                                                                                                                        const CameraParameters& camera_params)
            {
                  GpuMat  offsetMat; 
                std::vector<short3> offsetMat_host;

                for(short i= -1; i<=1; ++i)
                {
                       for(short j= -1; j<=1; ++j)
                       {
                            for(short k= -1;  k<=1; ++k)
                            {
                                offsetMat_host.push_back(make_short3(i,j,k));
                            }
                       } 
                }

                offsetMat = cv::cuda::createContinuous(1, offsetMat_host.size(), CV_16SC3);
                offsetMat.upload(cv::Mat(1, offsetMat_host.size(), CV_16SC3, offsetMat_host.data(), cv::Mat::AUTO_STEP));

                GpuMat  offsetMatMore; 
                std::vector<short3> offsetMatMore_host;

                for(short i= -2; i<=2; ++i)
                {
                       for(short j= -2; j<=2; ++j)
                       {
                            for(short k= -2;  k<=2; ++k)
                            {
                                offsetMatMore_host.push_back(make_short3(i,j,k));
                            }
                       } 
                }

                offsetMatMore = cv::cuda::createContinuous(1, offsetMatMore_host.size(), CV_16SC3);
                offsetMatMore.upload(cv::Mat(1, offsetMatMore_host.size(), CV_16SC3, offsetMatMore_host.data(), cv::Mat::AUTO_STEP));


                const dim3 threads(32, 32);
                const dim3 blocks((camera_params.image_width + threads.x - 1) / threads.x,
                                  (camera_params.image_height + threads.y - 1) / threads.y);

                extend_mesh_voxel<<<blocks, threads>>>(volume.uncertainty_volume, offsetMatMore, offsetMatMore_host.size() , volume.volume_size);

                mark_uncertainty_voxels<<<blocks, threads>>>(volume.uncertainty_volume, offsetMat, offsetMat_host.size() , volume.volume_size);
                return std::vector<Eigen::Vector3f>();
            }

            __global__
            void clear_candidate_points_cuda(
                PtrStepSz<short> uncertainty_volume,
                int3 volume_size
            )
            {
                const int x = blockIdx.x * blockDim.x + threadIdx.x;
                const int y = blockIdx.y * blockDim.y + threadIdx.y;

                if (x >= volume_size.x || y >= volume_size.y)
                    return;

                for(int z =0; z < volume_size.z; ++z )
                {
                    if(uncertainty_volume.ptr(z * volume_size.y + y)[x]==3)
                        uncertainty_volume.ptr(z * volume_size.y + y)[x] = 2;
                }
            }


            void clear_candidate_points(VolumeData& volume)
            {
                const dim3 threads(32, 32);
                const dim3 blocks((volume.volume_size.x + threads.x - 1) / threads.x,
                                  (volume.volume_size.y + threads.y - 1) / threads.y);
                
                clear_candidate_points_cuda<<<blocks, threads>>>(
                    volume.uncertainty_volume,
                    volume.volume_size
                );       
            }

            
        } 
    }
}
